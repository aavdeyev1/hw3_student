#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];
    // TODO, implement this kernel below
	
	//global thread ID's//for reading global mem
	int idX = blockIdx.x * blockDim.x + threadIdx.x;//column
	int IdY = blockIdx.y * blockDim.y + threadIdx.y;//row
	int mid_row =blockIdx.y + 1;
	int st_col = blockDim.x * blockIdx.x 
	int shared_row = 0;
	//make sure we are within bounds of input array.	
	if(idX >= width - 2 || idY >= width -2){
		return;
	}
	
	
	//read input to shared mem- > global
	//each thread has to read
	//if you are thread 0 or blockDim.x you have to do 
	  if(threadIdx.x == 0){
		  shared_row = 0;
		for(int i =mid_row - 1 ; i =< mid_row + 1; i++){			
			s_data[j*(blockDim.x+2) + threadIdx.x] = g_dataA[ st_col + (i*width) + threadIdx.x];
			shared_row++;
		}
	  }else if (threadIdx.x = blockDim.x || idX == width - 1){//
		  shared_row = 0;
		for(int i =mid_row - 1 ; i =< mid_row + 1; i++){			
			s_data[j*(blockDim.x+2)+ 2 + threadIdx.x] = g_dataA[ (st_col + 2) + (i*width) + threadIdx.x];
			shared_row++;
		}
	  }
	//everybody reads 3 elements
		shared_row = 0;
		for(int i =mid_row - 1 ; i =< mid_row + 1; i++){			
			s_data[j*(blockDim.x+2)+ 1 + threadIdx.x] = g_dataA[ (st_col + 1) + (i*width) + threadIdx.x];
			shared_row++;
		}
	
	
	
	__syncThreads();
	//sync threads
	
	
	
	
	//do the math
	//write the result
	
	
	
    
}


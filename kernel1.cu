#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) //float pitch is the number of float values in each row of the padded array...we need to offset the float ptch on each row
{
		// use width to disable threads out of array boundry. use floatpitch when doing global memory access.
    extern __shared__ float s_data[];
		s_data 
    // TODO, implement this kernel below
	
	//global thread ID's//for reading global mem
	int idX = blockIdx.x * blockDim.x + threadIdx.x + 1;//column
	int idY = blockIdx.y * blockDim.y + threadIdx.y + 1;//row
	int mid_row = blockIdx.y + 1;
	int st_col = blockDim.x * blockIdx.x; 
	int shared_row = 0;
	//make sure we are within bounds of input array.	
	if(idX >= (width - 1) || idY >= (width -1) || idX < 1 || idY < 1){
		return;
	}
	
	
	//read input to shared mem- > global
	//each thread has to read
	//if you are thread 0 or blockDim.x you have to do 
	  if(threadIdx.x == 0){
		  shared_row = 0;
			for(int i =mid_row - 1 ; i <= mid_row + 1; i++){			
				s_data[shared_row*(blockDim.x+2) + threadIdx.x] = g_dataA[ st_col + (i*floatpitch) + threadIdx.x];
				shared_row++;
			}
	  }
		if (idX == width-2) {
			shared_row = 0;
			for(int i = mid_row - 1 ; i <= mid_row + 1; i++){			
				s_data[shared_row*(blockDim.x+2)+ 2 + threadIdx.x] = g_dataA[ (st_col + 2) + (i*floatpitch) + threadIdx.x];
				shared_row++;
			}
		}
		else if (threadIdx.x == blockDim.x - 1){//
			shared_row = 0;
			for(int i = mid_row - 1 ; i <= mid_row + 1; i++){			
				s_data[shared_row*(blockDim.x+2)+ 2 + threadIdx.x] = g_dataA[ (st_col + 2) + (i*floatpitch) + threadIdx.x];
				shared_row++;
			}
	  }
	//everybody reads 3 elements
    shared_row = 0;
    for(int i = mid_row - 1 ; i <= mid_row + 1; i++){			
        s_data[shared_row*(blockDim.x+2)+ 1 + threadIdx.x] = g_dataA[ (st_col + 1) + (i*floatpitch) + threadIdx.x];
        shared_row++;
    }
	
	__syncthreads();
	//sync threads
    int s_rowwidth = blockDim.x +2;
	// //do the math
    g_dataB[st_col + 1 + threadIdx.x + floatpitch * mid_row] = (                                                        
    .1*(s_data[threadIdx.x])    +
    .1*(s_data[threadIdx.x+1])  +
    .1*(s_data[threadIdx.x+2])  +
    
    .1*(s_data[(s_rowwidth*1) + threadIdx.x])      +
    .2*(s_data[(s_rowwidth*1) + threadIdx.x + 1])  +
    .1*(s_data[(s_rowwidth*1) + threadIdx.x + 2])  +
    
    .1*(s_data[(s_rowwidth*2) + threadIdx.x])      +
    .1*(s_data[(s_rowwidth*2) + threadIdx.x + 1])  +
    .1*(s_data[(s_rowwidth*2) + threadIdx.x + 2])    ) * 0.95; 
	//write the result   
}


#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];
    // TODO, implement this kernel below
	
	//global thread ID's//for reading global mem
	int idX = blockIdx.x * blockDim.x + threadIdx.x;//column
	int idY = blockIdx.y * blockDim.y + threadIdx.y;//row
	int mid_row = blockIdx.y + 1;
	int st_col = blockDim.x * blockIdx.x; 
	int shared_row = 0;
	//make sure we are within bounds of input array.	
	if(idX >= width - 2 || idY >= width -2){
		return;
	}
	
	
	//read input to shared mem- > global
	//each thread has to read
	//if you are thread 0 or blockDim.x you have to do 
	  if(threadIdx.x == 0){
		  shared_row = 0;
		for(int i =mid_row - 1 ; i <= mid_row + 1; i++){			
			s_data[shared_row*(blockDim.x+2) + threadIdx.x] = g_dataA[ st_col + (i*width) + threadIdx.x];
			shared_row++;
		}
	  }
      else if (threadIdx.x == blockDim.x || idX == width - 1){//
		    shared_row = 0;
		    for(int i = mid_row - 1 ; i <= mid_row + 1; i++){			
			    s_data[shared_row*(blockDim.x+2)+ 2 + threadIdx.x] = g_dataA[ (st_col + 2) + (i*width) + threadIdx.x];
			    shared_row++;
		    }
	  }
	//everybody reads 3 elements
    shared_row = 0;
    for(int i =mid_row - 1 ; i <= mid_row + 1; i++){			
        s_data[shared_row*(blockDim.x+2)+ 1 + threadIdx.x] = g_dataA[ (st_col + 1) + (i*width) + threadIdx.x];
        shared_row++;
    }
	
	__syncthreads();
	//sync threads
    int s_rowwidth = blockDim.x +2;
	//do the math
    g_dataB[st_col + 1 + threadIdx.x + width * mid_row] = (                                                        
    .1*(s_data[threadIdx.x])    +
    .1*(s_data[threadIdx.x+1])  +
    .1*(s_data[threadIdx.x+2])  +
    
    .1*(s_data[(s_rowwidth*1) + threadIdx.x])      +
    .2*(s_data[(s_rowwidth*1) + threadIdx.x + 1])  +
    .1*(s_data[(s_rowwidth*1) + threadIdx.x + 2])  +
    
    .1*(s_data[(s_rowwidth*2) + threadIdx.x])      +
    .1*(s_data[(s_rowwidth*2) + threadIdx.x + 1])  +
    .1*(s_data[(s_rowwidth*2) + threadIdx.x + 2])    ) * 0.95; 
	//write the result
	
	
	
    
}

